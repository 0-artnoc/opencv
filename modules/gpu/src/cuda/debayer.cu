#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <opencv2/gpu/device/common.hpp>
#include <opencv2/gpu/device/vec_traits.hpp>

namespace cv { namespace gpu {
    namespace device
    {
        template <class SrcPtr, typename T>
        __global__ void Bayer2BGR(const SrcPtr src, PtrStep_<T> dst, const int width, const int height, const bool glob_blue_last, const bool glob_start_with_green)
        {
            const int tx = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y >= height)
                return;

            const bool blue_last = (y & 1) ? !glob_blue_last : glob_blue_last;
            const bool start_with_green = (y & 1) ? !glob_start_with_green : glob_start_with_green;

            int x = tx * 2;

            if (start_with_green)
            {
                --x;

                if (tx == 0)
                {
                    const int t0 = (src(y, 1) + src(y + 2, 1) + 1) >> 1;
                    const int t1 = (src(y + 1, 0) + src(y + 1, 2) + 1) >> 1;

                    T res;
                    res.x = blue_last ? t0 : t1;
                    res.y = src(y + 1, 1);
                    res.z = blue_last ? t1 : t0;

                    dst(y + 1, 0) = dst(y + 1, 1) = res;
                    if (y == 0)
                    {
                        dst(0, 0) = dst(0, 1) = res;
                    }
                    else if (y == height - 1)
                    {
                        dst(height + 1, 0) = dst(height + 1, 1) = res;
                    }
                }
            }

            if (x >= 0 && x <= width - 2)
            {
                const int t0 = (src(y, x) + src(y, x + 2) + src(y + 2, x) + src(y + 2, x + 2) + 2) >> 2;
                const int t1 = (src(y, x + 1) + src(y + 1, x) + src(y + 1, x + 2) + src(y + 2, x + 1) + 2) >> 2;

                const int t2 = (src(y, x + 2) + src(y + 2, x + 2) + 1) >> 1;
                const int t3 = (src(y + 1, x + 1) + src(y + 1, x + 3) + 1) >> 1;

                T res1, res2;

                if (blue_last)
                {
                    res1.x = t0;
                    res1.y = t1;
                    res1.z = src(y + 1, x + 1);

                    res2.x = t2;
                    res2.y = src(y + 1, x + 2);
                    res2.z = t3;
                }
                else
                {
                    res1.x = src(y + 1, x + 1);
                    res1.y = t1;
                    res1.z = t0;

                    res2.x = t3;
                    res2.y = src(y + 1, x + 2);
                    res2.z = t2;
                }

                dst(y + 1, x + 1) = res1;
                dst(y + 1, x + 2) = res2;

                if (y == 0)
                {
                    dst(0, x + 1) = res1;
                    dst(0, x + 2) = res2;

                    if (x == 0)
                    {
                        dst(0, 0) = res1;
                    }
                    else if (x == width - 2)
                    {
                        dst(0, width + 1) = res2;
                    }
                }
                else if (y == height - 1)
                {
                    dst(height + 1, x + 1) = res1;
                    dst(height + 1, x + 2) = res2;

                    if (x == 0)
                    {
                        dst(height + 1, 0) = res1;
                    }
                    else if (x == width - 2)
                    {
                        dst(height + 1, width + 1) = res2;
                    }
                }

                if (x == 0)
                {
                    dst(y + 1, 0) = res1;
                }
                else if (x == width - 2)
                {
                    dst(y + 1, width + 1) = res2;
                }
            }
            else if (x == width - 1)
            {
                const int t0 = (src(y, x) + src(y, x + 2) + src(y + 2, x) + src(y + 2, x + 2) + 2) >> 2;
                const int t1 = (src(y, x + 1) + src(y + 1, x) + src(y + 1, x + 2) + src(y + 2, x + 1) + 2) >> 2;

                T res;
                res.x = blue_last ? t0 : src(y + 1, x + 1);
                res.y = t1;
                res.z = blue_last ? src(y + 1, x + 1) : t0;

                dst(y + 1, x + 1) = dst(y + 1, x + 2) = res;
                if (y == 0)
                {
                    dst(0, x + 1) = dst(0, x + 2) = res;
                }
                else if (y == height - 1)
                {
                    dst(height + 1, x + 1) = dst(height + 1, x + 2) = res;
                }
            }
        }

        template <typename T, int cn>
        void Bayer2BGR_gpu(DevMem2Db src, DevMem2Db dst, bool blue_last, bool start_with_green, hipStream_t stream)
        {
            typedef typename TypeVec<T, cn>::vec_type dst_t;

            const int width = src.cols - 2;
            const int height = src.rows - 2;

            const dim3 total(divUp(width, 2), height);

            const dim3 block(32, 8);
            const dim3 grid(divUp(total.x, block.x), divUp(total.y, block.y));

            Bayer2BGR<PtrStep_<T>, dst_t><<<grid, block, 0, stream>>>((DevMem2D_<T>)src, (DevMem2D_<dst_t>)dst, width, height, blue_last, start_with_green);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        template void Bayer2BGR_gpu<uchar, 3>(DevMem2Db src, DevMem2Db dst, bool blue_last, bool start_with_green, hipStream_t stream);
        template void Bayer2BGR_gpu<uchar, 4>(DevMem2Db src, DevMem2Db dst, bool blue_last, bool start_with_green, hipStream_t stream);
        template void Bayer2BGR_gpu<ushort, 3>(DevMem2Db src, DevMem2Db dst, bool blue_last, bool start_with_green, hipStream_t stream);
        template void Bayer2BGR_gpu<ushort, 4>(DevMem2Db src, DevMem2Db dst, bool blue_last, bool start_with_green, hipStream_t stream);
    }
}}
