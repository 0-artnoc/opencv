#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <thrust/sort.h>
#include "opencv2/gpu/device/common.hpp"
#include "opencv2/gpu/device/emulation.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace hough
    {
        __device__ unsigned int g_counter;

        const int PIXELS_PER_THREAD = 16;

        __global__ void buildPointList(const DevMem2Db src, unsigned int* list)
        {
            __shared__ unsigned int s_queues[4][32 * PIXELS_PER_THREAD];
            __shared__ unsigned int s_qsize[4];
            __shared__ unsigned int s_start[4];

            const int x = blockIdx.x * blockDim.x * PIXELS_PER_THREAD + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y >= src.rows)
                return;

            if (threadIdx.x == 0)
                s_qsize[threadIdx.y] = 0;

            __syncthreads();

            // fill the queue
            for (int i = 0, xx = x; i < PIXELS_PER_THREAD && xx < src.cols; ++i, xx += blockDim.x)
            {
                if (src(y, xx))
                {
                    const unsigned int val = (y << 16) | xx;
                    int qidx = Emulation::smem::atomicInc(&s_qsize[threadIdx.y], (unsigned int)(-1));
                    s_queues[threadIdx.y][qidx] = val;
                }
            }

            __syncthreads();

            // let one thread reserve the space required in the global list
            if (threadIdx.x == 0 && threadIdx.y == 0)
            {
                // find how many items are stored in each list
                unsigned int total_size = 0;
                for (int i = 0; i < blockDim.y; ++i)
                {
                    s_start[i] = total_size;
                    total_size += s_qsize[i];
                }

                //calculate the offset in the global list
                const unsigned int global_offset = atomicAdd(&g_counter, total_size);
                for (int i = 0; i < blockDim.y; ++i)
                    s_start[i] += global_offset;
            }

            __syncthreads();

            // copy local queues to global queue
            const unsigned int qsize = s_qsize[threadIdx.y];
            for(int i = threadIdx.x; i < qsize; i += blockDim.x)
            {
                unsigned int val = s_queues[threadIdx.y][i];
                list[s_start[threadIdx.y] + i] = val;
            }
        }

        unsigned int buildPointList_gpu(DevMem2Db src, unsigned int* list)
        {
            void* counter_ptr;
            cudaSafeCall( hipGetSymbolAddress(&counter_ptr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counter_ptr, 0, sizeof(unsigned int)) );

            const dim3 block(32, 4);
            const dim3 grid(divUp(src.cols, block.x * PIXELS_PER_THREAD), divUp(src.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(buildPointList), hipFuncCachePreferShared) );

            buildPointList<<<grid, block>>>(src, list);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            unsigned int total_count;
            cudaSafeCall( hipMemcpy(&total_count, counter_ptr, sizeof(unsigned int), hipMemcpyDeviceToHost) );

            return total_count;
        }

        __global__ void linesAccum(const unsigned int* list, const unsigned int count, PtrStep_<unsigned int> accum,
                                   const float irho, const float theta, const int numrho)
        {
            extern __shared__ unsigned int smem[];

            for (int i = threadIdx.x; i < numrho; i += blockDim.x)
                smem[i] = 0;
            __syncthreads();

            const int n = blockIdx.x;
            const float ang = n * theta;

            float sin_ang;
            float cos_ang;
            sincosf(ang, &sin_ang, &cos_ang);

            const float tabSin = sin_ang * irho;
            const float tabCos = cos_ang * irho;

            for (int i = threadIdx.x; i < count; i += blockDim.x)
            {
                // read one element from global memory
                const unsigned int qvalue = list[i];
                const unsigned int x = (qvalue & 0x0000FFFF);
                const unsigned int y = (qvalue >> 16) & 0x0000FFFF;

                int r = __float2int_rn(x * tabCos + y * tabSin);
                r += (numrho - 1) / 2;

                Emulation::smem::atomicInc(&smem[r], (unsigned int)(-1));
            }
            __syncthreads();

            for (int i = threadIdx.x; i < numrho; i += blockDim.x)
                accum(n + 1, i + 1) = smem[i];
        }

        void linesAccum_gpu(const unsigned int* list, unsigned int count, DevMem2D_<unsigned int> accum, float rho, float theta)
        {
            const dim3 block(1024);
            const dim3 grid(accum.rows - 2);

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(linesAccum), hipFuncCachePreferShared) );

            size_t smem_size = (accum.cols - 2) * sizeof(unsigned int);

            linesAccum<<<grid, block, smem_size>>>(list, count, accum, 1.0f / rho, theta, accum.cols - 2);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        __global__ void linesGetResult(const DevMem2D_<unsigned int> accum, float2* out, int* voices, const int maxSize,
                                       const float threshold, const float theta, const float rho, const int numrho)
        {
            __shared__ unsigned int smem[8][32];

            int r = blockIdx.x * (blockDim.x - 2) + threadIdx.x;
            int n = blockIdx.y * (blockDim.y - 2) + threadIdx.y;

            if (r >= accum.cols || n >= accum.rows)
                return;

            smem[threadIdx.y][threadIdx.x] = accum(n, r);
            __syncthreads();

            r -= 1;
            n -= 1;

            if (threadIdx.x == 0 || threadIdx.x == blockDim.x - 1 || threadIdx.y == 0 || threadIdx.y == blockDim.y - 1 || r >= accum.cols - 2 || n >= accum.rows - 2)
                return;

            if (smem[threadIdx.y][threadIdx.x] > threshold &&
                smem[threadIdx.y][threadIdx.x] >  smem[threadIdx.y - 1][threadIdx.x] &&
                smem[threadIdx.y][threadIdx.x] >= smem[threadIdx.y + 1][threadIdx.x] &&
                smem[threadIdx.y][threadIdx.x] >  smem[threadIdx.y][threadIdx.x - 1] &&
                smem[threadIdx.y][threadIdx.x] >= smem[threadIdx.y][threadIdx.x + 1])
            {
                float radius = (r - (numrho - 1) * 0.5f) * rho;
                float angle = n * theta;

                const unsigned int ind = atomicInc(&g_counter, (unsigned int)(-1));
                if (ind < maxSize)
                {
                    out[ind] = make_float2(radius, angle);
                    voices[ind] = smem[threadIdx.y][threadIdx.x];
                }
            }
        }

        unsigned int linesGetResult_gpu(DevMem2D_<unsigned int> accum, float2* out, int* voices, unsigned int maxSize,
                                        float rho, float theta, float threshold, bool doSort)
        {
            void* counter_ptr;
            cudaSafeCall( hipGetSymbolAddress(&counter_ptr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counter_ptr, 0, sizeof(unsigned int)) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(accum.cols, block.x - 2), divUp(accum.rows, block.y - 2));

            linesGetResult<<<grid, block>>>(accum, out, voices, maxSize, threshold, theta, rho, accum.cols - 2);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            unsigned int total_count;
            cudaSafeCall( hipMemcpy(&total_count, counter_ptr, sizeof(unsigned int), hipMemcpyDeviceToHost) );

            total_count = ::min(total_count, maxSize);

            if (doSort && total_count > 0)
            {
                thrust::device_ptr<float2> out_ptr(out);
                thrust::device_ptr<int> voices_ptr(voices);
                thrust::sort_by_key(voices_ptr, voices_ptr + total_count, out_ptr, thrust::greater<int>());
            }

            return total_count;
        }
    }
}}}
