#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <thrust/sort.h>
#include "opencv2/gpu/device/common.hpp"
#include "opencv2/gpu/device/emulation.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace hough
    {
        __device__ int g_counter;

        ////////////////////////////////////////////////////////////////////////
        // buildPointList

        const int PIXELS_PER_THREAD = 16;

        __global__ void buildPointList(const DevMem2Db src, unsigned int* list)
        {
            __shared__ int s_queues[4][32 * PIXELS_PER_THREAD];
            __shared__ int s_qsize[4];
            __shared__ int s_start[4];

            const int x = blockIdx.x * blockDim.x * PIXELS_PER_THREAD + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y >= src.rows)
                return;

            if (threadIdx.x == 0)
                s_qsize[threadIdx.y] = 0;

            __syncthreads();

            // fill the queue
            for (int i = 0, xx = x; i < PIXELS_PER_THREAD && xx < src.cols; ++i, xx += blockDim.x)
            {
                if (src(y, xx))
                {
                    const unsigned int val = (y << 16) | xx;
                    const int qidx = Emulation::smem::atomicAdd(&s_qsize[threadIdx.y], 1);
                    s_queues[threadIdx.y][qidx] = val;
                }
            }

            __syncthreads();

            // let one thread reserve the space required in the global list
            if (threadIdx.x == 0 && threadIdx.y == 0)
            {
                // find how many items are stored in each list
                int total_size = 0;
                for (int i = 0; i < blockDim.y; ++i)
                {
                    s_start[i] = total_size;
                    total_size += s_qsize[i];
                }

                // calculate the offset in the global list
                const int global_offset = atomicAdd(&g_counter, total_size);
                for (int i = 0; i < blockDim.y; ++i)
                    s_start[i] += global_offset;
            }

            __syncthreads();

            // copy local queues to global queue
            const int qsize = s_qsize[threadIdx.y];
            for(int i = threadIdx.x; i < qsize; i += blockDim.x)
            {
                const unsigned int val = s_queues[threadIdx.y][i];
                list[s_start[threadIdx.y] + i] = val;
            }
        }

        int buildPointList_gpu(DevMem2Db src, unsigned int* list)
        {
            void* counter_ptr;
            cudaSafeCall( hipGetSymbolAddress(&counter_ptr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counter_ptr, 0, sizeof(int)) );

            const dim3 block(32, 4);
            const dim3 grid(divUp(src.cols, block.x * PIXELS_PER_THREAD), divUp(src.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(buildPointList), hipFuncCachePreferShared) );

            buildPointList<<<grid, block>>>(src, list);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int total_count;
            cudaSafeCall( hipMemcpy(&total_count, counter_ptr, sizeof(int), hipMemcpyDeviceToHost) );

            return total_count;
        }

        ////////////////////////////////////////////////////////////////////////
        // linesAccum

        __global__ void linesAccumGlobal(const unsigned int* list, const int count, PtrStepi accum, const float irho, const float theta, const int numrho)
        {
            const int n = blockIdx.x;
            const float ang = n * theta;

            float sin_ang;
            float cos_ang;
            sincosf(ang, &sin_ang, &cos_ang);

            const float tabSin = sin_ang * irho;
            const float tabCos = cos_ang * irho;

            for (int i = threadIdx.x; i < count; i += blockDim.x)
            {
                const unsigned int qvalue = list[i];

                const int x = (qvalue & 0x0000FFFF);
                const int y = (qvalue >> 16) & 0x0000FFFF;

                int r = __float2int_rn(x * tabCos + y * tabSin);
                r += (numrho - 1) / 2;

                ::atomicAdd(accum.ptr(n + 1) + r + 1, 1);
            }
        }

        __global__ void linesAccumShared(const unsigned int* list, const int count, PtrStepi accum, const float irho, const float theta, const int numrho)
        {
            extern __shared__ int smem[];

            for (int i = threadIdx.x; i < numrho + 1; i += blockDim.x)
                smem[i] = 0;

            __syncthreads();

            const int n = blockIdx.x;
            const float ang = n * theta;

            float sin_ang;
            float cos_ang;
            sincosf(ang, &sin_ang, &cos_ang);

            const float tabSin = sin_ang * irho;
            const float tabCos = cos_ang * irho;

            for (int i = threadIdx.x; i < count; i += blockDim.x)
            {
                const unsigned int qvalue = list[i];

                const int x = (qvalue & 0x0000FFFF);
                const int y = (qvalue >> 16) & 0x0000FFFF;

                int r = __float2int_rn(x * tabCos + y * tabSin);
                r += (numrho - 1) / 2;

                Emulation::smem::atomicAdd(&smem[r + 1], 1);
            }

            __syncthreads();

            for (int i = threadIdx.x; i < numrho; i += blockDim.x)
                accum(n + 1, i) = smem[i];
        }

        void linesAccum_gpu(const unsigned int* list, int count, DevMem2Di accum, float rho, float theta, size_t sharedMemPerBlock, bool has20)
        {
            const dim3 block(has20 ? 1024 : 512);
            const dim3 grid(accum.rows - 2);

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(linesAccumShared), hipFuncCachePreferShared) );

            size_t smemSize = (accum.cols - 1) * sizeof(int);

            if (smemSize < sharedMemPerBlock - 1000)
                linesAccumShared<<<grid, block, smemSize>>>(list, count, accum, 1.0f / rho, theta, accum.cols - 2);
            else
                linesAccumGlobal<<<grid, block>>>(list, count, accum, 1.0f / rho, theta, accum.cols - 2);

            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////
        // linesGetResult

        __global__ void linesGetResult(const DevMem2Di accum, float2* out, int* votes, const int maxSize, const float threshold, const float theta, const float rho, const int numrho)
        {
            __shared__ int smem[8][32];

            int r = blockIdx.x * (blockDim.x - 2) + threadIdx.x;
            int n = blockIdx.y * (blockDim.y - 2) + threadIdx.y;

            if (r >= accum.cols || n >= accum.rows)
                return;

            smem[threadIdx.y][threadIdx.x] = accum(n, r);
            __syncthreads();

            r -= 1;
            n -= 1;

            if (threadIdx.x == 0 || threadIdx.x == blockDim.x - 1 || threadIdx.y == 0 || threadIdx.y == blockDim.y - 1 || r >= accum.cols - 2 || n >= accum.rows - 2)
                return;

            if (smem[threadIdx.y][threadIdx.x] > threshold &&
                smem[threadIdx.y][threadIdx.x] >  smem[threadIdx.y - 1][threadIdx.x] &&
                smem[threadIdx.y][threadIdx.x] >= smem[threadIdx.y + 1][threadIdx.x] &&
                smem[threadIdx.y][threadIdx.x] >  smem[threadIdx.y][threadIdx.x - 1] &&
                smem[threadIdx.y][threadIdx.x] >= smem[threadIdx.y][threadIdx.x + 1])
            {
                const float radius = (r - (numrho - 1) * 0.5f) * rho;
                const float angle = n * theta;

                const int ind = ::atomicAdd(&g_counter, 1);
                if (ind < maxSize)
                {
                    out[ind] = make_float2(radius, angle);
                    votes[ind] = smem[threadIdx.y][threadIdx.x];
                }
            }
        }

        int linesGetResult_gpu(DevMem2Di accum, float2* out, int* votes, int maxSize, float rho, float theta, float threshold, bool doSort)
        {
            void* counter_ptr;
            cudaSafeCall( hipGetSymbolAddress(&counter_ptr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counter_ptr, 0, sizeof(int)) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(accum.cols, block.x - 2), divUp(accum.rows, block.y - 2));

            linesGetResult<<<grid, block>>>(accum, out, votes, maxSize, threshold, theta, rho, accum.cols - 2);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int total_count;
            cudaSafeCall( hipMemcpy(&total_count, counter_ptr, sizeof(int), hipMemcpyDeviceToHost) );

            total_count = ::min(total_count, maxSize);

            if (doSort && total_count > 0)
            {
                thrust::device_ptr<float2> out_ptr(out);
                thrust::device_ptr<int> votes_ptr(votes);
                thrust::sort_by_key(votes_ptr, votes_ptr + total_count, out_ptr, thrust::greater<int>());
            }

            return total_count;
        }
    }
}}}
