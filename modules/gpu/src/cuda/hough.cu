#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <thrust/sort.h>
#include "opencv2/gpu/device/common.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace hough
    {
        __global__ void linesAccum(const DevMem2Db src, PtrStep_<uint> accum, const float theta, const int numangle, const int numrho, const float irho)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x >= src.cols || y >= src.rows)
                return;

            if (src(y, x))
            {
                float ang = 0.0f;
                for(int n = 0; n < numangle; ++n, ang += theta)
                {
                    float sin_ang;
                    float cos_ang;
                    sincosf(ang, &sin_ang, &cos_ang);

                    const float tabSin = sin_ang * irho;
                    const float tabCos = cos_ang * irho;

                    int r = __float2int_rn(x * tabCos + y * tabSin);
                    r += (numrho - 1) / 2;

                    atomicInc(accum.ptr(n + 1) + r + 1, (unsigned int)-1);
                }
            }
        }

        void linesAccum_gpu(DevMem2Db src, PtrStep_<uint> accum, float theta, int numangle, int numrho, float irho)
        {
            const dim3 block(32, 8);
            const dim3 grid(divUp(src.cols, block.x), divUp(src.rows, block.y));

            linesAccum<<<grid, block>>>(src, accum, theta, numangle, numrho, irho);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        __device__ unsigned int g_counter;

        __global__ void linesGetResult(const DevMem2D_<uint> accum, float2* out, int* voices, const int maxSize, const float threshold, const float theta, const float rho, const int numrho)
        {
            __shared__ uint smem[8][32];

            int r = blockIdx.x * (blockDim.x - 2) + threadIdx.x;
            int n = blockIdx.y * (blockDim.y - 2) + threadIdx.y;

            if (r >= accum.cols || n >= accum.rows)
                return;

            smem[threadIdx.y][threadIdx.x] = accum(n, r);
            __syncthreads();

            r -= 1;
            n -= 1;

            if (threadIdx.x == 0 || threadIdx.x == blockDim.x - 1 || threadIdx.y == 0 || threadIdx.y == blockDim.y - 1 || r >= accum.cols - 2 || n >= accum.rows - 2)
                return;

            if (smem[threadIdx.y][threadIdx.x] > threshold &&
                smem[threadIdx.y][threadIdx.x] >  smem[threadIdx.y - 1][threadIdx.x] &&
                smem[threadIdx.y][threadIdx.x] >= smem[threadIdx.y + 1][threadIdx.x] &&
                smem[threadIdx.y][threadIdx.x] >  smem[threadIdx.y][threadIdx.x - 1] &&
                smem[threadIdx.y][threadIdx.x] >= smem[threadIdx.y][threadIdx.x + 1])
            {
                float radius = (r - (numrho - 1) * 0.5f) * rho;
                float angle = n * theta;

                const unsigned int ind = atomicInc(&g_counter, (unsigned int)(-1));
                if (ind < maxSize)
                {
                    out[ind] = make_float2(radius, angle);
                    voices[ind] = smem[threadIdx.y][threadIdx.x];
                }
            }
        }

        int linesGetResult_gpu(DevMem2D_<uint> accum, float2* out, int* voices, int maxSize, float threshold, float theta, float rho, bool doSort)
        {
            void* counter_ptr;
            cudaSafeCall( hipGetSymbolAddress(&counter_ptr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counter_ptr, 0, sizeof(unsigned int)) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(accum.cols, block.x - 2), divUp(accum.rows, block.y - 2));

            linesGetResult<<<grid, block>>>(accum, out, voices, maxSize, threshold, theta, rho, accum.cols - 2);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            uint total_count;
            cudaSafeCall( hipMemcpy(&total_count, counter_ptr, sizeof(uint), hipMemcpyDeviceToHost) );

            if (doSort)
            {
                thrust::device_ptr<float2> out_ptr(out);
                thrust::device_ptr<int> voices_ptr(voices);
                thrust::sort_by_key(voices_ptr, voices_ptr + total_count, out_ptr, thrust::greater<int>());
            }

            return total_count;
        }
    }
}}}
