#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or bpied warranties, including, but not limited to, the bpied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <thrust/sort.h>
#include "opencv2/gpu/device/common.hpp"
#include "opencv2/gpu/device/emulation.hpp"

namespace cv { namespace gpu { namespace device
{
    namespace hough
    {
        __device__ int g_counter;

        ////////////////////////////////////////////////////////////////////////
        // buildPointList

        const int PIXELS_PER_THREAD = 16;

        __global__ void buildPointList(const DevMem2Db src, unsigned int* list)
        {
            __shared__ unsigned int s_queues[4][32 * PIXELS_PER_THREAD];
            __shared__ int s_qsize[4];
            __shared__ int s_globStart[4];

            const int x = blockIdx.x * blockDim.x * PIXELS_PER_THREAD + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (y >= src.rows)
                return;

            if (threadIdx.x == 0)
                s_qsize[threadIdx.y] = 0;

            __syncthreads();

            // fill the queue
            const uchar* srcRow = src.ptr(y);
            for (int i = 0, xx = x; i < PIXELS_PER_THREAD && xx < src.cols; ++i, xx += blockDim.x)
            {
                if (srcRow[xx])
                {
                    const unsigned int val = (y << 16) | xx;
                    const int qidx = Emulation::smem::atomicAdd(&s_qsize[threadIdx.y], 1);
                    s_queues[threadIdx.y][qidx] = val;
                }
            }

            __syncthreads();

            // let one thread reserve the space required in the global list
            if (threadIdx.x == 0 && threadIdx.y == 0)
            {
                // find how many items are stored in each list
                int totalSize = 0;
                for (int i = 0; i < blockDim.y; ++i)
                {
                    s_globStart[i] = totalSize;
                    totalSize += s_qsize[i];
                }

                // calculate the offset in the global list
                const int globalOffset = atomicAdd(&g_counter, totalSize);
                for (int i = 0; i < blockDim.y; ++i)
                    s_globStart[i] += globalOffset;
            }

            __syncthreads();

            // copy local queues to global queue
            const int qsize = s_qsize[threadIdx.y];
            int gidx = s_globStart[threadIdx.y] + threadIdx.x;
            for(int i = threadIdx.x; i < qsize; i += blockDim.x, gidx += blockDim.x)
                list[gidx] = s_queues[threadIdx.y][i];
        }

        int buildPointList_gpu(DevMem2Db src, unsigned int* list)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(32, 4);
            const dim3 grid(divUp(src.cols, block.x * PIXELS_PER_THREAD), divUp(src.rows, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(buildPointList), hipFuncCachePreferShared) );

            buildPointList<<<grid, block>>>(src, list);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            return totalCount;
        }

        ////////////////////////////////////////////////////////////////////////
        // linesAccum

        __global__ void linesAccumGlobal(const unsigned int* list, const int count, PtrStepi accum, const float irho, const float theta, const int numrho)
        {
            const int n = blockIdx.x;
            const float ang = n * theta;

            float sinVal;
            float cosVal;
            sincosf(ang, &sinVal, &cosVal);
            sinVal *= irho;
            cosVal *= irho;

            const int shift = (numrho - 1) / 2;

            int* accumRow = accum.ptr(n + 1);
            for (int i = threadIdx.x; i < count; i += blockDim.x)
            {
                const unsigned int val = list[i];

                const int x = (val & 0xFFFF);
                const int y = (val >> 16) & 0xFFFF;

                int r = __float2int_rn(x * cosVal + y * sinVal);
                r += shift;

                ::atomicAdd(accumRow + r + 1, 1);
            }
        }

        __global__ void linesAccumShared(const unsigned int* list, const int count, PtrStepi accum, const float irho, const float theta, const int numrho)
        {
            extern __shared__ int smem[];

            for (int i = threadIdx.x; i < numrho + 1; i += blockDim.x)
                smem[i] = 0;

            __syncthreads();

            const int n = blockIdx.x;
            const float ang = n * theta;

            float sinVal;
            float cosVal;
            sincosf(ang, &sinVal, &cosVal);
            sinVal *= irho;
            cosVal *= irho;

            const int shift = (numrho - 1) / 2;

            for (int i = threadIdx.x; i < count; i += blockDim.x)
            {
                const unsigned int val = list[i];

                const int x = (val & 0xFFFF);
                const int y = (val >> 16) & 0xFFFF;

                int r = __float2int_rn(x * cosVal + y * sinVal);
                r += shift;

                Emulation::smem::atomicAdd(&smem[r + 1], 1);
            }

            __syncthreads();

            int* accumRow = accum.ptr(n + 1);
            for (int i = threadIdx.x; i < numrho + 1; i += blockDim.x)
                accumRow[i] = smem[i];
        }

        void linesAccum_gpu(const unsigned int* list, int count, DevMem2Di accum, float rho, float theta, size_t sharedMemPerBlock, bool has20)
        {
            const dim3 block(has20 ? 1024 : 512);
            const dim3 grid(accum.rows - 2);

            size_t smemSize = (accum.cols - 1) * sizeof(int);

            if (smemSize < sharedMemPerBlock - 1000)
                linesAccumShared<<<grid, block, smemSize>>>(list, count, accum, 1.0f / rho, theta, accum.cols - 2);
            else
                linesAccumGlobal<<<grid, block>>>(list, count, accum, 1.0f / rho, theta, accum.cols - 2);

            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////
        // linesGetResult

        __global__ void linesGetResult(const DevMem2Di accum, float2* out, int* votes, const int maxSize, const float rho, const float theta, const float threshold, const int numrho)
        {
            const int r = blockIdx.x * blockDim.x + threadIdx.x;
            const int n = blockIdx.y * blockDim.y + threadIdx.y;

            if (r >= accum.cols - 2 && n >= accum.rows - 2)
                return;

            const int curVotes = accum(n + 1, r + 1);

            if (curVotes > threshold &&
                curVotes >  accum(n + 1, r) &&
                curVotes >= accum(n + 1, r + 2) &&
                curVotes >  accum(n, r + 1) &&
                curVotes >= accum(n + 2, r + 1))
            {
                const float radius = (r - (numrho - 1) * 0.5f) * rho;
                const float angle = n * theta;

                const int ind = ::atomicAdd(&g_counter, 1);
                if (ind < maxSize)
                {
                    out[ind] = make_float2(radius, angle);
                    votes[ind] = curVotes;
                }
            }
        }

        int linesGetResult_gpu(DevMem2Di accum, float2* out, int* votes, int maxSize, float rho, float theta, float threshold, bool doSort)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(accum.cols - 2, block.x), divUp(accum.rows - 2, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(linesGetResult), hipFuncCachePreferL1) );

            linesGetResult<<<grid, block>>>(accum, out, votes, maxSize, rho, theta, threshold, accum.cols - 2);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            totalCount = ::min(totalCount, maxSize);

            if (doSort && totalCount > 0)
            {
                thrust::device_ptr<float2> outPtr(out);
                thrust::device_ptr<int> votesPtr(votes);
                thrust::sort_by_key(votesPtr, votesPtr + totalCount, outPtr, thrust::greater<int>());
            }

            return totalCount;
        }

        ////////////////////////////////////////////////////////////////////////
        // circlesAccumCenters

        __global__ void circlesAccumCenters(const unsigned int* list, const int count, const PtrStepi dx, const PtrStepi dy,
                                            PtrStepi accum, const int width, const int height, const int minRadius, const int maxRadius, const float idp)
        {
            const int SHIFT = 10;
            const int ONE = 1 << SHIFT;

            const int tid = blockIdx.x * blockDim.x + threadIdx.x;

            if (tid >= count)
                return;

            const unsigned int val = list[tid];

            const int x = (val & 0xFFFF);
            const int y = (val >> 16) & 0xFFFF;

            const int vx = dx(y, x);
            const int vy = dy(y, x);

            if (vx == 0 && vy == 0)
                return;

            const float mag = ::sqrtf(vx * vx + vy * vy);

            const int x0 = __float2int_rn((x * idp) * ONE);
            const int y0 = __float2int_rn((y * idp) * ONE);

            int sx = __float2int_rn((vx * idp) * ONE / mag);
            int sy = __float2int_rn((vy * idp) * ONE / mag);

            // Step from minRadius to maxRadius in both directions of the gradient
            for (int k1 = 0; k1 < 2; ++k1)
            {
                int x1 = x0 + minRadius * sx;
                int y1 = y0 + minRadius * sy;

                for (int r = minRadius; r <= maxRadius; x1 += sx, y1 += sy, ++r)
                {
                    const int x2 = x1 >> SHIFT;
                    const int y2 = y1 >> SHIFT;

                    if (x2 < 0 || x2 >= width || y2 < 0 || y2 >= height)
                        break;

                    ::atomicAdd(accum.ptr(y2 + 1) + x2 + 1, 1);
                }

                sx = -sx;
                sy = -sy;
            }
        }

        void circlesAccumCenters_gpu(const unsigned int* list, int count, PtrStepi dx, PtrStepi dy, DevMem2Di accum, int minRadius, int maxRadius, float idp)
        {
            const dim3 block(256);
            const dim3 grid(divUp(count, block.x));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(circlesAccumCenters), hipFuncCachePreferL1) );

            circlesAccumCenters<<<grid, block>>>(list, count, dx, dy, accum, accum.cols - 2, accum.rows - 2, minRadius, maxRadius, idp);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );
        }

        ////////////////////////////////////////////////////////////////////////
        // buildCentersList

        __global__ void buildCentersList(const DevMem2Di accum, unsigned int* centers, const int threshold)
        {
            const int x = blockIdx.x * blockDim.x + threadIdx.x;
            const int y = blockIdx.y * blockDim.y + threadIdx.y;

            if (x < accum.cols - 2 && y < accum.rows - 2)
            {
                const int top = accum(y, x + 1);

                const int left = accum(y + 1, x);
                const int cur = accum(y + 1, x + 1);
                const int right = accum(y + 1, x + 2);

                const int bottom = accum(y + 2, x + 1);

                if (cur > threshold && cur > top && cur >= bottom && cur >  left && cur >= right)
                {
                    const unsigned int val = (y << 16) | x;
                    const int idx = ::atomicAdd(&g_counter, 1);
                    centers[idx] = val;
                }
            }
        }

        int buildCentersList_gpu(DevMem2Di accum, unsigned int* centers, int threshold)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(32, 8);
            const dim3 grid(divUp(accum.cols - 2, block.x), divUp(accum.rows - 2, block.y));

            cudaSafeCall( hipFuncSetCacheConfig(reinterpret_cast<const void*>(buildCentersList), hipFuncCachePreferL1) );

            buildCentersList<<<grid, block>>>(accum, centers, threshold);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            return totalCount;
        }

        ////////////////////////////////////////////////////////////////////////
        // circlesAccumRadius

        __global__ void circlesAccumRadius(const unsigned int* centers, const unsigned int* list, const int count,
                                           float3* circles, const int maxCircles, const float dp,
                                           const int minRadius, const int maxRadius, const int histSize, const int threshold)
        {
            extern __shared__ int smem[];

            for (int i = threadIdx.x; i < histSize + 2; i += blockDim.x)
                smem[i] = 0;
            __syncthreads();

            unsigned int val = centers[blockIdx.x];

            float cx = (val & 0xFFFF);
            float cy = (val >> 16) & 0xFFFF;

            cx = (cx + 0.5f) * dp;
            cy = (cy + 0.5f) * dp;

            for (int i = threadIdx.x; i < count; i += blockDim.x)
            {
                val = list[i];

                const int x = (val & 0xFFFF);
                const int y = (val >> 16) & 0xFFFF;

                const float rad = ::sqrtf((cx - x) * (cx - x) + (cy - y) * (cy - y));
                if (rad >= minRadius && rad <= maxRadius)
                {
                    const int r = __float2int_rn(rad - minRadius);

                    Emulation::smem::atomicAdd(&smem[r + 1], 1);
                }
            }

            __syncthreads();

            for (int i = threadIdx.x; i < histSize; i += blockDim.x)
            {
                const int curVotes = smem[i + 1];

                if (curVotes >= threshold && curVotes > smem[i] && curVotes >= smem[i + 2])
                {
                    const int ind = ::atomicAdd(&g_counter, 1);
                    if (ind < maxCircles)
                        circles[ind] = make_float3(cx, cy, i + minRadius);
                }
            }
        }

        int circlesAccumRadius_gpu(const unsigned int* centers, int centersCount, const unsigned int* list, int count,
                                   float3* circles, int maxCircles, float dp, int minRadius, int maxRadius, int threshold, bool has20)
        {
            void* counterPtr;
            cudaSafeCall( hipGetSymbolAddress(&counterPtr, HIP_SYMBOL(g_counter)) );

            cudaSafeCall( hipMemset(counterPtr, 0, sizeof(int)) );

            const dim3 block(has20 ? 1024 : 512);
            const dim3 grid(centersCount);

            const int histSize = ::ceil(maxRadius - minRadius + 1);
            size_t smemSize = (histSize + 2) * sizeof(int);

            circlesAccumRadius<<<grid, block, smemSize>>>(centers, list, count, circles, maxCircles, dp, minRadius, maxRadius, histSize, threshold);
            cudaSafeCall( hipGetLastError() );

            cudaSafeCall( hipDeviceSynchronize() );

            int totalCount;
            cudaSafeCall( hipMemcpy(&totalCount, counterPtr, sizeof(int), hipMemcpyDeviceToHost) );

            totalCount = ::min(totalCount, maxCircles);

            return totalCount;
        }
    }
}}}
