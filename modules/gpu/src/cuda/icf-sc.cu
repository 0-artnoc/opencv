#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2008-2012, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <opencv2/gpu/device/common.hpp>
#include <icf.hpp>
#include <stdio.h>
#include <float.h>

namespace cv { namespace gpu { namespace device {
namespace icf {

    // ToDo: use textures or ancached load instruction.
    __global__ void magToHist(const uchar* __restrict__ mag,
                              const float* __restrict__ angle, const int angPitch,
                                    uchar* __restrict__ hog,   const int hogPitch, const int fh)
    {
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;

        const int bin = (int)(angle[y * angPitch + x]);
        const uchar val = mag[y * hogPitch + x];
        hog[((fh * bin) + y) * hogPitch + x] = val;
    }

    void fillBins(cv::gpu::PtrStepSzb hogluv, const cv::gpu::PtrStepSzf& nangle,
                  const int fw,  const int fh, const int bins, hipStream_t stream )
    {
        const uchar* mag = (const uchar*)hogluv.ptr(fh * bins);
        uchar* hog = (uchar*)hogluv.ptr();
        const float* angle = (const float*)nangle.ptr();

        dim3 block(32, 8);
        dim3 grid(fw / 32, fh / 8);

        magToHist<<<grid, block, 0, stream>>>(mag, angle, nangle.step / sizeof(float), hog, hogluv.step, fh);
        if (!stream)
        {
            cudaSafeCall( hipGetLastError() );
            cudaSafeCall( hipDeviceSynchronize() );
        }
    }

    template<typename Policy>
    struct PrefixSum
    {
    __device static void apply(float& impact)
        {
    #if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 300
    #pragma unroll
            // scan on shuffl functions
            for (int i = 1; i < Policy::WARP; i *= 2)
            {
                const float n = __shfl_up(impact, i, Policy::WARP);

                if (threadIdx.x >= i)
                    impact += n;
            }
    #else
            __shared__ volatile float ptr[Policy::STA_X * Policy::STA_Y];

            const int idx = threadIdx.y * Policy::STA_X + threadIdx.x;

            ptr[idx] = impact;

            if ( threadIdx.x >=  1) ptr [idx ] = (ptr [idx -  1] + ptr [idx]);
            if ( threadIdx.x >=  2) ptr [idx ] = (ptr [idx -  2] + ptr [idx]);
            if ( threadIdx.x >=  4) ptr [idx ] = (ptr [idx -  4] + ptr [idx]);
            if ( threadIdx.x >=  8) ptr [idx ] = (ptr [idx -  8] + ptr [idx]);
            if ( threadIdx.x >= 16) ptr [idx ] = (ptr [idx - 16] + ptr [idx]);

            impact = ptr[idx];
    #endif
        }
    };

    texture<int,  hipTextureType2D, hipReadModeElementType> thogluv;

    template<bool isUp>
    __device__ __forceinline__ float rescale(const Level& level, Node& node)
    {
        uchar4& scaledRect = node.rect;
        float relScale = level.relScale;
        float farea = (scaledRect.z - scaledRect.x) * (scaledRect.w - scaledRect.y);

        // rescale
        scaledRect.x = __float2int_rn(relScale * scaledRect.x);
        scaledRect.y = __float2int_rn(relScale * scaledRect.y);
        scaledRect.z = __float2int_rn(relScale * scaledRect.z);
        scaledRect.w = __float2int_rn(relScale * scaledRect.w);

        float sarea = (scaledRect.z - scaledRect.x) * (scaledRect.w - scaledRect.y);

        const float expected_new_area = farea * relScale * relScale;
        float approx = (sarea == 0)? 1: __fdividef(sarea, expected_new_area);

        float rootThreshold = (node.threshold & 0x0FFFFFFFU) * approx * level.scaling[(node.threshold >> 28) > 6];

        return rootThreshold;
    }

    template<>
    __device__ __forceinline__ float rescale<true>(const Level& level, Node& node)
    {
        uchar4& scaledRect = node.rect;
        float relScale = level.relScale;
        float farea = scaledRect.z * scaledRect.w;

        // rescale
        scaledRect.x = __float2int_rn(relScale * scaledRect.x);
        scaledRect.y = __float2int_rn(relScale * scaledRect.y);
        scaledRect.z = __float2int_rn(relScale * scaledRect.z);
        scaledRect.w = __float2int_rn(relScale * scaledRect.w);

        float sarea = scaledRect.z * scaledRect.w;

        const float expected_new_area = farea * relScale * relScale;
        float approx = __fdividef(sarea, expected_new_area);

        float rootThreshold = (node.threshold & 0x0FFFFFFFU) * approx * level.scaling[(node.threshold >> 28) > 6];

        return rootThreshold;
    }

    template<bool isUp>
    __device__ __forceinline__ int get(int x, int y, uchar4 area)
    {
        int a = tex2D(thogluv, x + area.x, y + area.y);
        int b = tex2D(thogluv, x + area.z, y + area.y);
        int c = tex2D(thogluv, x + area.z, y + area.w);
        int d = tex2D(thogluv, x + area.x, y + area.w);

        return (a - b + c - d);
    }

    template<>
    __device__ __forceinline__ int get<true>(int x, int y, uchar4 area)
    {
        x += area.x;
        y += area.y;
        int a = tex2D(thogluv, x, y);
        int b = tex2D(thogluv, x + area.z, y);
        int c = tex2D(thogluv, x + area.z, y + area.w);
        int d = tex2D(thogluv, x, y + area.w);

        return (a - b + c - d);
    }

    texture<float2,  hipTextureType2D, hipReadModeElementType> troi;

template<typename Policy>
template<bool isUp>
__device void CascadeInvoker<Policy>::detect(Detection* objects, const uint ndetections, uint* ctr, const int downscales) const
{
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x;

    // load Lavel
    __shared__ Level level;

    // check POI
    __shared__ volatile char roiCache[Policy::STA_Y];

    if (!threadIdx.y && !threadIdx.x)
        ((float2*)roiCache)[threadIdx.x] = tex2D(troi, blockIdx.y, x);

    __syncthreads();

    if (!roiCache[threadIdx.y]) return;

    if (!threadIdx.x)
        level = levels[downscales + blockIdx.z];

    if(x >= level.workRect.x || y >= level.workRect.y) return;

    int st = level.octave * level.step;
    const int stEnd = st + level.step;

    const int hogluvStep = gridDim.y * Policy::STA_Y;
    float confidence = 0.f;
    for(; st < stEnd; st += Policy::WARP)
    {
        const int nId = (st + threadIdx.x) * 3;

        Node node = nodes[nId];

        float threshold = rescale<isUp>(level, node);
        int sum = get<isUp>(x, y + (node.threshold >> 28) * hogluvStep, node.rect);

        int next = 1 + (int)(sum >= threshold);

        node = nodes[nId + next];
        threshold = rescale<isUp>(level, node);
        sum = get<isUp>(x, y + (node.threshold >> 28) * hogluvStep, node.rect);

        const int lShift = (next - 1) * 2 + (int)(sum >= threshold);
        float impact = leaves[(st + threadIdx.x) * 4 + lShift];

        PrefixSum<Policy>::apply(impact);
        confidence += impact;

        if(__any((confidence <= stages[(st + threadIdx.x)]))) st += 2048;
    }

    if(!threadIdx.x && st == stEnd &&  ((confidence - FLT_EPSILON) >= 0))
    {
        int idx = atomicInc(ctr, ndetections);
        objects[idx] = Detection(__float2int_rn(x * Policy::SHRINKAGE),
            __float2int_rn(y * Policy::SHRINKAGE), level.objSize.x, level.objSize.y, confidence);
    }
}

template<typename Policy, bool isUp>
__global__ void soft_cascade(const CascadeInvoker<Policy> invoker, Detection* objects, const uint n, uint* ctr, const int downs)
{
    invoker.template detect<isUp>(objects, n, ctr, downs);
}

template<typename Policy>
void CascadeInvoker<Policy>::operator()(const PtrStepSzb& roi, const PtrStepSzi& hogluv,
    PtrStepSz<uchar4> objects, PtrStepSzi counter, const int downscales, const hipStream_t& stream) const
{
    int fw = roi.rows;
    int fh = roi.cols;

    dim3 grid(fw, fh / Policy::STA_Y, downscales);

    uint* ctr = (uint*)(counter.ptr(0));
    Detection* det = (Detection*)objects.ptr();
    uint max_det = objects.cols / sizeof(Detection);

    hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
    cudaSafeCall( hipBindTexture2D(0, thogluv, hogluv.data, desc, hogluv.cols, hogluv.rows, hogluv.step));

    hipChannelFormatDesc desc_roi = hipCreateChannelDesc<typename Policy::roi_type>();
    cudaSafeCall( hipBindTexture2D(0, troi, roi.data, desc_roi, roi.cols / Policy::STA_Y, roi.rows, roi.step));

    const CascadeInvoker<Policy> inv = *this;

    soft_cascade<Policy, false><<<grid, Policy::block(), 0, stream>>>(inv, det, max_det, ctr, 0);
    cudaSafeCall( hipGetLastError());

    grid = dim3(fw, fh / Policy::STA_Y, scales - downscales);
    soft_cascade<Policy, true><<<grid, Policy::block(), 0, stream>>>(inv, det, max_det, ctr, downscales);

    if (!stream)
    {
        cudaSafeCall( hipGetLastError());
        cudaSafeCall( hipDeviceSynchronize());
    }
}

template void CascadeInvoker<GK107PolicyX4>::operator()(const PtrStepSzb& roi, const PtrStepSzi& hogluv,
    PtrStepSz<uchar4> objects, PtrStepSzi counter, const int downscales, const hipStream_t& stream) const;

}
}}}