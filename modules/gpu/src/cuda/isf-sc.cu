#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2008-2012, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <icf.hpp>
#include <opencv2/gpu/device/saturate_cast.hpp>

namespace cv { namespace gpu {


 namespace device {

enum {
    HOG_BINS = 6,
    HOG_LUV_BINS = 10,
    WIDTH = 640,
    HEIGHT = 480,
    GREY_OFFSET = HEIGHT * HOG_LUV_BINS
};

/* Returns the nearest upper power of two, works only for
the typical GPU thread count (pert block) values */
int power_2up(unsigned int n)
{
    if (n < 1) return 1;
    else if (n < 2) return 2;
    else if (n < 4) return 4;
    else if (n < 8) return 8;
    else if (n < 16) return 16;
    else if (n < 32) return 32;
    else if (n < 64) return 64;
    else if (n < 128) return 128;
    else if (n < 256) return 256;
    else if (n < 512) return 512;
    else if (n < 1024) return 1024;
    return -1; // Input is too big
}


__device__ __forceinline__ uchar grey(const uchar3 rgb)
{
    return saturate_cast<uchar>(rgb.x * 0.114f + rgb.y * 0.587f + rgb.z * 0.299f);
}

__device__ __forceinline__ void luv(const uchar3 rgb, uchar& l, uchar& u, uchar& v)
{

}

__global__ void rgb2grayluv(const uchar3* __restrict__ rgb, uchar* __restrict__ hog,
                            const int rgbPitch, const int hogPitch)
{
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    const uchar3 color = rgb[rgbPitch * y + x];

    uchar l, u, v;
    luv(color, l, u, v);

    hog[hogPitch *  y + x] = l;
    hog[hogPitch * (y + HEIGHT) + x] = u;
    hog[hogPitch * (y + 2 * HEIGHT) + x] = v;
    hog[hogPitch * (y + 3 * HEIGHT) + x] = grey(color);
}

__device__ __forceinline__
int qangle(const float &y, const float &x)
{
    int bin = 0;
//     const float2 &bin_vector_zero = const_angle_bins_vectors[0];
//     float max_dot_product = fabs(x*bin_vector_zero.x + y*bin_vector_zero.y);

//     // let us hope this gets unrolled
// #pragma unroll
//     for(int i=1; i < num_angles_bin; i+=1)
//     {
//         const float2 &bin_vector_i = const_angle_bins_vectors[i];
//         //const float2 bin_vector_i = const_angle_bins_vectors[i];
//         //const float2 &bin_vector_i = angle_bins_vectors[i];
//         const float dot_product = fabs(x*bin_vector_i.x + y*bin_vector_i.y);
//         if(dot_product > max_dot_product)
//         {
//             max_dot_product = dot_product;
//             index = i;
//         }
//     }

    return bin;
}

// texture<uchar, 2, hipReadModeElementType> tgray;
__global__ void gray2hog(const uchar* __restrict__ gray, uchar* __restrict__ hog, const int pitch, const float norm)
{
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    // derivative
    float dx = gray[y * pitch + x + 1];
    dx -= gray[y * pitch + x - 1];

    float dy = gray[(y + 1) * pitch + x];
    dy -= gray[(y -1) * pitch + x - 1];

    // mag and angle
    const uchar mag =  saturate_cast<uchar>(sqrtf(dy * dy + dx * dx) * norm);
    const int bin = qangle(dx, dy);

}

template <int FACTOR>
__device__ __forceinline__ uchar shrink(const uchar* ptr, const int pitch, const int y, const int x)
{
    int out = 0;
#pragma unroll
    for(int dy = 0; dy < FACTOR; ++dy)
#pragma unroll
        for(int dx = 0; dx < FACTOR; ++dx)
        {
            out += ptr[dy * pitch + dx];
        }

    return saturate_cast<uchar>(out / FACTOR);
}

template<int FACTOR>
__global__ void decimate(const uchar* __restrict__ hogluv, uchar* __restrict__ shrank,
                        const int inPitch, const int outPitch )
{
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    const uchar* ptr = hogluv + (FACTOR * y) * inPitch + (FACTOR * x);

    shrank[ y * outPitch + x]= shrink<FACTOR>(ptr, inPitch, y, x);
}

__global__ void intRow(const uchar* __restrict__ hogluv, ushort* __restrict__ sum,
                       const int inPitch, const int outPitch)
{

}

__global__ void intCol(ushort* __restrict__ sum, const int pitch)
{

}


__global__ void detect(const cv::gpu::icf::Cascade cascade, const uchar* __restrict__ hogluv, const int pitch)
{
    cascade.detectAt();
}

}

void __device icf::Cascade::detectAt() const
{

}

void icf::Cascade::detect(const cv::gpu::PtrStepSzb& hogluv, hipStream_t stream) const
{
    // detection kernel
    dim3 block(32, 8, 1);
    dim3 grid(32 * ChannelStorage::FRAME_WIDTH / 32, ChannelStorage::FRAME_HEIGHT / 8, 64);
    device::detect<<<grid, block, 0, stream>>>(*this, hogluv, hogluv.step / sizeof(ushort));
    if (!stream)
        cudaSafeCall( hipDeviceSynchronize() );

}

void icf::ChannelStorage::frame(const cv::gpu::PtrStepSz<uchar3>& rgb, hipStream_t stream)
{
    // color convertin kernel
    dim3 block(32, 8);
    dim3 grid(FRAME_WIDTH / 32, FRAME_HEIGHT / 8);

    uchar * channels = (uchar*)dmem.ptr(FRAME_HEIGHT * HOG_BINS);
    device::rgb2grayluv<<<grid, block, 0, stream>>>((uchar3*)rgb.ptr(), channels,
                                                    rgb.step / sizeof(uchar3), dmem.step);
    cudaSafeCall( hipGetLastError());

    // hog calculation kernel
    channels = (uchar*)dmem.ptr(FRAME_HEIGHT * HOG_LUV_BINS);
    device::gray2hog<<<grid, block, 0, stream>>>(channels, (uchar*)dmem.ptr(), dmem.step, magnitudeScaling);
    cudaSafeCall( hipGetLastError() );

    const int shrWidth  = FRAME_WIDTH / shrinkage;
    const int shrHeight = FRAME_HEIGHT / shrinkage;

    // decimate kernel
    grid = dim3(shrWidth / 32, shrHeight / 8);
    device::decimate<4><<<grid, block, 0, stream>>>((uchar*)dmem.ptr(), (uchar*)shrunk.ptr(), dmem.step, shrunk.step);
    cudaSafeCall( hipGetLastError() );

    // integrate rows
    block = dim3(shrWidth, 1);
    grid = dim3(shrHeight * HOG_LUV_BINS, 1);
    device::intRow<<<grid, block, 0, stream>>>((uchar*)shrunk.ptr(), (ushort*)hogluv.ptr(),
        shrunk.step, hogluv.step / sizeof(ushort));
    cudaSafeCall( hipGetLastError() );

    // integrate cols
    block = dim3(128, 1);
    grid = dim3(shrWidth * HOG_LUV_BINS, 1);
    device::intCol<<<grid, block, 0, stream>>>((ushort*)hogluv.ptr(), hogluv.step / hogluv.step / sizeof(ushort));
    cudaSafeCall( hipGetLastError() );
}

}}