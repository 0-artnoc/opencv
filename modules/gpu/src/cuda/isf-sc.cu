#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2008-2012, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <icf.hpp>
#include <opencv2/gpu/device/saturate_cast.hpp>
#include <stdio.h>
#include <float.h>

//#define LOG_CUDA_CASCADE

#if defined LOG_CUDA_CASCADE
# define dprintf(format, ...) \
            do { printf(format, __VA_ARGS__); } while (0)
#else
# define dprintf(format, ...)
#endif

namespace cv { namespace gpu { namespace device {

namespace icf {

    enum {
        HOG_BINS = 6,
        HOG_LUV_BINS = 10,
        WIDTH = 640,
        HEIGHT = 480,
        GREY_OFFSET = HEIGHT * HOG_LUV_BINS
    };

    __global__ void magToHist(const uchar* __restrict__ mag,
                              const float* __restrict__ angle, const int angPitch,
                                    uchar* __restrict__ hog,   const int hogPitch)
    {
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;

        const int bin = (int)(angle[y * angPitch + x]);
        const uchar val = mag[y * angPitch + x];

        hog[((HEIGHT * bin) + y) * hogPitch + x] = val;
    }

    void fillBins(cv::gpu::PtrStepSzb hogluv, const cv::gpu::PtrStepSzf& nangle)
    {
        const uchar* mag = (const uchar*)hogluv.ptr(HEIGHT * HOG_BINS);
        uchar* hog = (uchar*)hogluv.ptr();
        const float* angle = (const float*)nangle.ptr();

        dim3 block(32, 8);
        dim3 grid(WIDTH / 32, HEIGHT / 8);

        magToHist<<<grid, block>>>(mag, angle, nangle.step / sizeof(float), hog, hogluv.step);
        cudaSafeCall( hipGetLastError() );
        cudaSafeCall( hipDeviceSynchronize() );
    }
}

__global__ void detect(const cv::gpu::icf::Cascade cascade, const int* __restrict__ hogluv, const int pitch,
    PtrStepSz<uchar4> objects)
{
    cascade.detectAt(hogluv, pitch, objects);
}

}

float __device icf::Cascade::rescale(const icf::Level& level, uchar4& scaledRect,
                                     const int channel, const float threshold) const
{
    dprintf("feature %d box %d %d %d %d\n", channel, scaledRect.x, scaledRect.y, scaledRect.z, scaledRect.w);
    dprintf("rescale: %f [%f %f]\n",level.relScale, level.scaling[0], level.scaling[1]);

    float relScale = level.relScale;
    float farea = (scaledRect.z - scaledRect.x) * (scaledRect.w - scaledRect.y);

    // rescale
    scaledRect.x = __float2int_rn(relScale * scaledRect.x);
    scaledRect.y = __float2int_rn(relScale * scaledRect.y);
    scaledRect.z = __float2int_rn(relScale * scaledRect.z);
    scaledRect.w = __float2int_rn(relScale * scaledRect.w);

    float sarea = (scaledRect.z - scaledRect.x) * (scaledRect.w - scaledRect.y);


    float approx = 1.f;
    if (fabs(farea - 0.f) > FLT_EPSILON && fabs(farea - 0.f) > FLT_EPSILON)
    {
        const float expected_new_area = farea * relScale * relScale;
        approx = expected_new_area / sarea;
    }

    dprintf("new rect: %d box %d %d %d %d  rel areas %f %f\n", channel,
        scaledRect.x, scaledRect.y, scaledRect.z, scaledRect.w, farea * relScale * relScale, sarea);

    // compensation areas rounding
    float rootThreshold = threshold / approx;
    // printf("    approx %f\n", rootThreshold);
    rootThreshold *= level.scaling[(int)(channel > 6)];

    dprintf("approximation %f %f -> %f %f\n", approx, threshold, rootThreshold, level.scaling[(int)(channel > 6)]);

    return rootThreshold;
}

typedef unsigned char uchar;
float __device get(const int* __restrict__ hogluv, const int pitch,
                   const int x, const int y, int channel, uchar4 area)
{
    dprintf("feature box %d %d %d %d ", area.x, area.y, area.z, area.w);
    dprintf("get for channel %d\n", channel);
    dprintf("extract feature for: [%d %d] [%d %d] [%d %d] [%d %d]\n",
        x + area.x, y + area.y,  x + area.z, y + area.y,  x + area.z,y + area.w,
        x + area.x, y + area.w);
    dprintf("at point %d %d with offset %d\n", x, y, 0);

    const int* curr = hogluv + ((channel * 121) + y) * pitch;

    int a = curr[area.y * pitch + x + area.x];
    int b = curr[area.y * pitch + x + area.z];
    int c = curr[area.w * pitch + x + area.z];
    int d = curr[area.w * pitch + x + area.x];

    dprintf("    retruved integral values: %d %d %d %d\n", a, b, c, d);

    return (a - b + c - d);
}


void __device icf::Cascade::detectAt(const int* __restrict__ hogluv, const int pitch,
                                    PtrStepSz<uchar4>& objects) const
{
    const icf::Level* lls = (const icf::Level*)levels.ptr();

    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    // if (x > 0 || y > 0) return;

    Level level = lls[blockIdx.z];
    if (x >= level.workRect.x || y >= level.workRect.y) return;

    dprintf("level: %d (%f %f) [%f %f] (%d %d) (%d %d)\n", level.octave, level.relScale, level.shrScale,
        level.scaling[0], level.scaling[1], level.workRect.x, level.workRect.y, level.objSize.x, level.objSize.y);

    const Octave octave = ((const Octave*)octaves.ptr())[level.octave];
    // printf("Octave: %d %d %d (%d %d) %f\n", octave.index, octave.stages,
    //     octave.shrinkage, octave.size.x, octave.size.y, octave.scale);

    const int stBegin = octave.index * octave.stages, stEnd = stBegin + octave.stages;

    float detectionScore = 0.f;

    int st = stBegin;
    for(; st < stEnd; ++st)
    {
        const float stage = stages(0, st);
        dprintf("Stage: %f\n", stage);
        {
            const int nId = st * 3;

            // work with root node
            const Node node = ((const Node*)nodes.ptr())[nId];

            dprintf("Node: %d %f\n", node.feature, node.threshold);

            const Feature feature = ((const Feature*)features.ptr())[node.feature];

            uchar4 scaledRect = feature.rect;
            float threshold = rescale(level, scaledRect, feature.channel, node.threshold);

            float sum = get(hogluv,pitch, x, y, feature.channel, scaledRect);

            dprintf("root feature %d %f\n",feature.channel, sum);

            int next = 1 + (int)(sum >= threshold);

            dprintf("go: %d (%f >= %f)\n\n" ,next, sum, threshold);

            // leaves
            const Node leaf = ((const Node*)nodes.ptr())[nId + next];
            const Feature fLeaf = ((const Feature*)features.ptr())[leaf.feature];

            scaledRect = fLeaf.rect;
            threshold = rescale(level, scaledRect, fLeaf.channel, leaf.threshold);
            sum = get(hogluv, pitch, x, y, fLeaf.channel, scaledRect);

            const int lShift = (next - 1) * 2 + (int)(sum >= threshold);
            float impact = leaves(0, (st * 4) + lShift);

            detectionScore += impact;

            dprintf("decided: %d (%f >= %f) %d %f\n\n" ,next, sum, threshold, lShift, impact);
            dprintf("extracted stage:\n");
            dprintf("ct %f\n", stage);
            dprintf("computed score %f\n\n", detectionScore);
            dprintf("\n\n");
        }

        if (detectionScore <= stage || st - stBegin == 100) break;
    }

    dprintf("x %d y %d: %d\n", x, y, st - stBegin);

    if (st == stEnd)
    {
        uchar4 a;
        a.x = level.workRect.x;
        a.y = level.workRect.y;
        objects(0, threadIdx.x) = a;
    }
}

void icf::Cascade::detect(const cv::gpu::PtrStepSzi& hogluv, PtrStepSz<uchar4> objects, hipStream_t stream) const
{
    dim3 block(32, 8, 1);
    dim3 grid(ChannelStorage::FRAME_WIDTH / 32, ChannelStorage::FRAME_HEIGHT / 8, 47);
    device::detect<<<grid, block, 0, stream>>>(*this, hogluv, hogluv.step / sizeof(int), objects);
    cudaSafeCall( hipGetLastError() );
    if (!stream)
        cudaSafeCall( hipDeviceSynchronize() );
}

}}