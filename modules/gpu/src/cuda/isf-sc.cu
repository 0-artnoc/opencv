#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2008-2012, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <icf.hpp>
#include <opencv2/gpu/device/saturate_cast.hpp>
#include <stdio.h>
#include <float.h>

namespace cv { namespace gpu { namespace device {

namespace icf {

    enum {
        HOG_BINS = 6,
        HOG_LUV_BINS = 10,
        WIDTH = 640,
        HEIGHT = 480,
        GREY_OFFSET = HEIGHT * HOG_LUV_BINS
    };

    __global__ void magToHist(const uchar* __restrict__ mag,
                              const float* __restrict__ angle, const int angPitch,
                                    uchar* __restrict__ hog,   const int hogPitch)
    {
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;

        const int bin = (int)(angle[y * angPitch + x]);
        const uchar val = mag[y * angPitch + x];

        hog[((HEIGHT * bin) + y) * hogPitch + x] = val;
    }

    void fillBins(cv::gpu::PtrStepSzb hogluv, const cv::gpu::PtrStepSzf& nangle)
    {
        const uchar* mag = (const uchar*)hogluv.ptr(HEIGHT * HOG_BINS);
        uchar* hog = (uchar*)hogluv.ptr();
        const float* angle = (const float*)nangle.ptr();

        dim3 block(32, 8);
        dim3 grid(WIDTH / 32, HEIGHT / 8);

        magToHist<<<grid, block>>>(mag, angle, nangle.step / sizeof(float), hog, hogluv.step);
        cudaSafeCall( hipGetLastError() );
        cudaSafeCall( hipDeviceSynchronize() );
    }
}

enum {
    HOG_BINS = 6,
    HOG_LUV_BINS = 10,
    WIDTH = 640,
    HEIGHT = 480,
    GREY_OFFSET = HEIGHT * HOG_LUV_BINS
};

/* Returns the nearest upper power of two, works only for
the typical GPU thread count (pert block) values */
int power_2up(unsigned int n)
{
    if (n < 1) return 1;
    else if (n < 2) return 2;
    else if (n < 4) return 4;
    else if (n < 8) return 8;
    else if (n < 16) return 16;
    else if (n < 32) return 32;
    else if (n < 64) return 64;
    else if (n < 128) return 128;
    else if (n < 256) return 256;
    else if (n < 512) return 512;
    else if (n < 1024) return 1024;
    return -1; // Input is too big
}


__device__ __forceinline__ uchar grey(const uchar3 rgb)
{
    return saturate_cast<uchar>(rgb.x * 0.114f + rgb.y * 0.587f + rgb.z * 0.299f);
}

__device__ __forceinline__ void luv(const uchar3 rgb, uchar& l, uchar& u, uchar& v)
{

}

__global__ void rgb2grayluv(const uchar3* __restrict__ rgb, uchar* __restrict__ hog,
                            const int rgbPitch, const int hogPitch)
{
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    const uchar3 color = rgb[rgbPitch * y + x];

    uchar l, u, v;
    luv(color, l, u, v);

    hog[hogPitch *  y + x] = l;
    hog[hogPitch * (y + HEIGHT) + x] = u;
    hog[hogPitch * (y + 2 * HEIGHT) + x] = v;
    hog[hogPitch * (y + 3 * HEIGHT) + x] = grey(color);
}

__device__ __forceinline__
int qangle(const float &y, const float &x)
{
    int bin = 0;
//     const float2 &bin_vector_zero = const_angle_bins_vectors[0];
//     float max_dot_product = fabs(x*bin_vector_zero.x + y*bin_vector_zero.y);

//     // let us hope this gets unrolled
// #pragma unroll
//     for(int i=1; i < num_angles_bin; i+=1)
//     {
//         const float2 &bin_vector_i = const_angle_bins_vectors[i];
//         //const float2 bin_vector_i = const_angle_bins_vectors[i];
//         //const float2 &bin_vector_i = angle_bins_vectors[i];
//         const float dot_product = fabs(x*bin_vector_i.x + y*bin_vector_i.y);
//         if(dot_product > max_dot_product)
//         {
//             max_dot_product = dot_product;
//             index = i;
//         }
//     }

    return bin;
}

// texture<uchar, 2, hipReadModeElementType> tgray;
__global__ void gray2hog(const uchar* __restrict__ gray, uchar* __restrict__ hog, const int pitch, const float norm)
{
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    // derivative
    float dx = gray[y * pitch + x + 1];
    dx -= gray[y * pitch + x - 1];

    float dy = gray[(y + 1) * pitch + x];
    dy -= gray[(y -1) * pitch + x - 1];

    // mag and angle
    const uchar mag =  saturate_cast<uchar>(sqrtf(dy * dy + dx * dx) * norm);
    const int bin = qangle(dx, dy);

}

template <int FACTOR>
__device__ __forceinline__ uchar shrink(const uchar* ptr, const int pitch, const int y, const int x)
{
    int out = 0;
#pragma unroll
    for(int dy = 0; dy < FACTOR; ++dy)
#pragma unroll
        for(int dx = 0; dx < FACTOR; ++dx)
        {
            out += ptr[dy * pitch + dx];
        }

    return saturate_cast<uchar>(out / FACTOR);
}

template<int FACTOR>
__global__ void decimate(const uchar* __restrict__ hogluv, uchar* __restrict__ shrank,
                        const int inPitch, const int outPitch )
{
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    const uchar* ptr = hogluv + (FACTOR * y) * inPitch + (FACTOR * x);

    shrank[ y * outPitch + x]= shrink<FACTOR>(ptr, inPitch, y, x);
}

__global__ void intRow(const uchar* __restrict__ hogluv, ushort* __restrict__ sum,
                       const int inPitch, const int outPitch)
{

}

__global__ void intCol(ushort* __restrict__ sum, const int pitch)
{

}


__global__ void detect(const cv::gpu::icf::Cascade cascade, const uchar* __restrict__ hogluv, const int pitch,
    PtrStepSz<uchar4> objects)
{
    cascade.detectAt(hogluv, pitch, objects);
}

}

float __device icf::Cascade::rescale(const icf::Level& level, uchar4& scaledRect,
                                     const int channel, const float threshold) const
{
    float relScale = level.relScale;
    float farea = (scaledRect.z - scaledRect.x) * (scaledRect.w - scaledRect.y);

    // rescale
    scaledRect.x = __float2int_rn(relScale * scaledRect.x);
    scaledRect.y = __float2int_rn(relScale * scaledRect.y);
    scaledRect.z = __float2int_rn(relScale * scaledRect.z);
    scaledRect.w = __float2int_rn(relScale * scaledRect.w);

    float sarea = (scaledRect.z - scaledRect.x) * (scaledRect.w - scaledRect.y);

    float approx = 1.f;
    if (fabs(farea - 0.f) > FLT_EPSILON && fabs(farea - 0.f) > FLT_EPSILON)
    {
        const float expected_new_area = farea * relScale * relScale;
        approx = expected_new_area / sarea;
    }

    // compensation areas rounding
    float rootThreshold = threshold / approx;
    rootThreshold *= level.scaling[(int)(channel > 6)];

    return rootThreshold;
}

typedef unsigned char uchar;
float __device get(const uchar* __restrict__ hogluv, const int pitch,
                   const int x, const int y, int channel, uchar4 area)
{
    const uchar* curr = hogluv + ((channel * 121) + y) * pitch;

    int a = curr[area.y * pitch + x + area.x];
    int b = curr[area.y * pitch + x + area.z];
    int c = curr[area.w * pitch + x + area.z];
    int d = curr[area.w * pitch + x + area.x];

    return (a - b + c - d);
}


void __device icf::Cascade::detectAt(const uchar* __restrict__ hogluv, const int pitch,
                                    PtrStepSz<uchar4>& objects) const
{
    const icf::Level* lls = (const icf::Level*)levels.ptr();
    Level level = lls[0];

    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= level.workRect.x || y >= level.workRect.y) return;

    const Octave octave = ((const Octave*)octaves.ptr())[level.octave];
    const int stBegin = octave.index * octave.stages, stEnd = stBegin + octave.stages;

    float detectionScore = 0.f;

    int st = stBegin;
    for(; st < stEnd; ++st)
    {
        const float stage = stages(0, st);
        {
            const int nId = st * 3;

            // work with root node
            const Node node = ((const Node*)nodes.ptr())[nId];
            const Feature feature = ((const Feature*)features.ptr())[node.feature];

            uchar4 scaledRect = feature.rect;
            float threshold = rescale(level, scaledRect, feature.channel, node.threshold);

            float sum = get(hogluv,pitch, x, y, feature.channel, scaledRect);

            int next = 1 + (int)(sum >= threshold);

            // leaves
            const Node leaf = ((const Node*)nodes.ptr())[nId + next];
            const Feature fLeaf = ((const Feature*)features.ptr())[leaf.feature];

            scaledRect = fLeaf.rect;
            threshold = rescale(level, scaledRect, feature.channel, node.threshold);
            sum = get(hogluv, pitch, x, y, fLeaf.channel, scaledRect);

            const int lShift = (next - 1) * 2 + (int)(sum >= threshold);
            float impact = leaves(0, (st * 4) + lShift);

            detectionScore += impact;
        }

        if (detectionScore <= stage) break;
    }

    // if (!threadIdx.x && !threadIdx.y)// printf("%f %d\n", detectionScore, st);
    //     printf("x %d y %d: %d\n", x, y, st);

    if (st == stEnd)
    {
        // printf("  got %d\n", st);
        uchar4 a;
        a.x = level.workRect.x;
        a.y = level.workRect.y;
        objects(0, threadIdx.x) = a;
    }
}

void icf::Cascade::detect(const cv::gpu::PtrStepSzb& hogluv, PtrStepSz<uchar4> objects,
                          hipStream_t stream) const
{
    // detection kernel
    dim3 block(32, 8, 1);
    // dim3 grid(32 * ChannelStorage::FRAME_WIDTH / 32, ChannelStorage::FRAME_HEIGHT / 8, 1);
    dim3 grid(ChannelStorage::FRAME_WIDTH / 32, ChannelStorage::FRAME_HEIGHT / 8, 1);
    device::detect<<<grid, block, 0, stream>>>(*this, hogluv, hogluv.step / sizeof(ushort), objects);
    cudaSafeCall( hipGetLastError() );
    if (!stream)
        cudaSafeCall( hipDeviceSynchronize() );

}

////////////////////////////////////////////////////



void icf::ChannelStorage::frame(const cv::gpu::PtrStepSz<uchar3>& rgb, hipStream_t stream)
{
//     // // color convertin kernel
//     // dim3 block(32, 8);
//     // dim3 grid(FRAME_WIDTH / 32, FRAME_HEIGHT / 8);

//     // uchar * channels = (uchar*)dmem.ptr(FRAME_HEIGHT * HOG_BINS);
//     // device::rgb2grayluv<<<grid, block, 0, stream>>>((uchar3*)rgb.ptr(), channels,
//     //                                                 rgb.step / sizeof(uchar3), dmem.step);
//     // cudaSafeCall( hipGetLastError());

//     // // hog calculation kernel
//     // channels = (uchar*)dmem.ptr(FRAME_HEIGHT * HOG_LUV_BINS);
//     // device::gray2hog<<<grid, block, 0, stream>>>(channels, (uchar*)dmem.ptr(), dmem.step, magnitudeScaling);
//     // cudaSafeCall( hipGetLastError() );

//     // const int shrWidth  = FRAME_WIDTH / shrinkage;
//     // const int shrHeight = FRAME_HEIGHT / shrinkage;

//     // // decimate kernel
//     // grid = dim3(shrWidth / 32, shrHeight / 8);
//     // device::decimate<4><<<grid, block, 0, stream>>>((uchar*)dmem.ptr(), (uchar*)shrunk.ptr(), dmem.step, shrunk.step);
//     // cudaSafeCall( hipGetLastError() );

//     // // integrate rows
//     // block = dim3(shrWidth, 1);
//     // grid = dim3(shrHeight * HOG_LUV_BINS, 1);
//     // device::intRow<<<grid, block, 0, stream>>>((uchar*)shrunk.ptr(), (ushort*)hogluv.ptr(),
//     //     shrunk.step, hogluv.step / sizeof(ushort));
//     // cudaSafeCall( hipGetLastError() );

//     // // integrate cols
//     // block = dim3(128, 1);
//     // grid = dim3(shrWidth * HOG_LUV_BINS, 1);
//     // device::intCol<<<grid, block, 0, stream>>>((ushort*)hogluv.ptr(), hogluv.step / hogluv.step / sizeof(ushort));
//     // cudaSafeCall( hipGetLastError() );
}

}}